#include "hip/hip_runtime.h"
/**
 * @file saxpysingle.cu
 *
 * CUDA implementation of S = a*X + y
 */

#include <cstdlib>
#include <cmath>
#include <iostream>

#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "../benchmark_suite.hpp"

#ifndef _SIZE_
#error The _SIZE_ constant is not defined!
#endif

void pre_saxpysingle(void);
void run_saxpysinglecublas(void);
void run_mysaxpysingle(void);
void post_saxpysingle(void);

#define PERR(a) p_cudaerr(#a, a)
#define PSTAT(a) p_cublasstat(#a, a)

static void p_cudaerr(const char *msg, hipError_t err)
{
    if (err != hipSuccess)
    	std::cerr << "cudaerr: " << msg << ": " << hipGetErrorString(err) << std::endl;
}

static void p_cublasstat(const char *msg, hipblasStatus_t stat)
{
#   define p_case(v) \
	       case v: \
           std::cerr << "cublasstat: " << msg << " returned " << #v << std::endl; \
           break

	switch (stat) {
	p_case(HIPBLAS_STATUS_NOT_INITIALIZED);
	p_case(HIPBLAS_STATUS_ALLOC_FAILED);
	p_case(HIPBLAS_STATUS_INVALID_VALUE);
	p_case(HIPBLAS_STATUS_ARCH_MISMATCH);
	p_case(HIPBLAS_STATUS_MAPPING_ERROR);
	p_case(HIPBLAS_STATUS_EXECUTION_FAILED);
	p_case(HIPBLAS_STATUS_INTERNAL_ERROR);
	p_case(HIPBLAS_STATUS_NOT_SUPPORTED);
	p_case(HIPBLAS_STATUS_UNKNOWN);
	default:
		break;
	}
#undef p_case
}

static double a_scalar;
static double y_value;
static double *s_arr = NULL;
static double *x_arr = NULL;

int main(void)
{
	std::cout << "[<<<< BENCHMARKING CUBLAS IMPLEMENTATION >>>>]" << std::endl;
	BENCHMARK(pre_saxpysingle, run_saxpysinglecublas, post_saxpysingle);
	std::cout << "[<<<<<<<<<<<<<<<<<<<<<<>>>>>>>>>>>>>>>>>>>>>>]" << std::endl;
	std::cout << std::endl << std::endl;
	std::cout << "[<<<< BENCHMARKING MY OWN IMPLEMENTATION >>>>]" << std::endl;
	BENCHMARK(pre_saxpysingle, run_mysaxpysingle, post_saxpysingle);
	std::cout << "[<<<<<<<<<<<<<<<<<<<<<<>>>>>>>>>>>>>>>>>>>>>>]" << std::endl;
	return 0;
}

void pre_saxpysingle(void)
{
	s_arr = new double[_SIZE_];
	x_arr = new double[_SIZE_];
	for (int i = 0; i < _SIZE_; ++i) {
		x_arr[i] = ((double) (i % 101)) / (43.8);
	}
	a_scalar = 11.1;
	y_value = 103.5;
}

void post_saxpysingle(void)
{
	delete[] s_arr;
	delete[] x_arr;
}

void run_saxpysinglecublas(void)
{
	double a;
	double *d_x;
	double *d_y;
	hipblasHandle_t handle;

	PERR(hipMalloc(&d_x, (_SIZE_) * sizeof(double)));
	PERR(hipMalloc(&d_y, sizeof(double)));
	PSTAT(hipblasCreate(&handle));
	PSTAT(hipblasSetVector(_SIZE_, sizeof(double), x_arr, 1, d_x, 1));
	PSTAT(hipblasSetVector(1, sizeof(double), &y_value, 1, d_y, 1));
	a = a_scalar;

	PSTAT(hipblasDaxpy(handle, _SIZE_, &a, d_y, 0, d_x, 1));

	PSTAT(hipblasGetVector(_SIZE_, sizeof(double), d_x, 1, s_arr, 1));
	hipblasDestroy(handle);
	hipFree(d_x);
	hipFree(d_y);
}



// My own implementation
__global__ void my_saxpysingle(int n, double a, double *x, double y)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < n)
		x[i] = (a * x[i]) + y;
}

void run_mysaxpysingle(void)
{
	double a;
	double y;
	double *d_x;
	int threadsPerBlock;
	int numBlocks;

	PERR(hipDeviceGetAttribute(&threadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, 0));
	numBlocks = ((_SIZE_) + threadsPerBlock - 1) / threadsPerBlock;

	PERR(hipMalloc(&d_x, (_SIZE_) * sizeof(double)));
	PERR(hipMemcpy(d_x, x_arr, (_SIZE_) * sizeof(double), hipMemcpyHostToDevice));
	a = a_scalar;
	y = y_value;

	my_saxpysingle<<<numBlocks,threadsPerBlock>>>(_SIZE_, a, d_x, y);
	PERR(hipDeviceSynchronize());

	PERR(hipMemcpy(s_arr, d_x, (_SIZE_) * sizeof(double), hipMemcpyDeviceToHost));
	hipFree(d_x);
}
