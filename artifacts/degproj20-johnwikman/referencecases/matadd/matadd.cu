/**
 * @file matadd.cu
 *
 * CUDA (cuBLAS) implementation of S = A + B
 */

#include <cstdlib>
#include <cmath>
#include <iostream>

#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "../benchmark_suite.hpp"

#ifndef _SIZE_
#error The _SIZE_ constant is not defined!
#endif

void pre_matadd(void);
void run_matadd(void);
void post_matadd(void);

#define PERR(a) p_cudaerr(#a, a)
#define PSTAT(a) p_cublasstat(#a, a)

static void p_cudaerr(const char *msg, hipError_t err)
{
    if (err != hipSuccess)
    	std::cerr << "cudaerr: " << msg << ": " << hipGetErrorString(err) << std::endl;
}

static void p_cublasstat(const char *msg, hipblasStatus_t stat)
{
#   define p_case(v) \
	       case v: \
           std::cerr << "cublasstat: " << msg << " returned " << #v << std::endl; \
           break

	switch (stat) {
	p_case(HIPBLAS_STATUS_NOT_INITIALIZED);
	p_case(HIPBLAS_STATUS_ALLOC_FAILED);
	p_case(HIPBLAS_STATUS_INVALID_VALUE);
	p_case(HIPBLAS_STATUS_ARCH_MISMATCH);
	p_case(HIPBLAS_STATUS_MAPPING_ERROR);
	p_case(HIPBLAS_STATUS_EXECUTION_FAILED);
	p_case(HIPBLAS_STATUS_INTERNAL_ERROR);
	p_case(HIPBLAS_STATUS_NOT_SUPPORTED);
	p_case(HIPBLAS_STATUS_UNKNOWN);
	default:
		break;
	}
#undef p_case
}

static double *A_mat = NULL;
static double *B_mat = NULL;
static double *S_mat = NULL;

int main(void)
{
	BENCHMARK(pre_matadd, run_matadd, post_matadd);
	return 0;
}

void pre_matadd(void)
{
	A_mat = new double[(_SIZE_) * (_SIZE_)];
	B_mat = new double[(_SIZE_) * (_SIZE_)];
	S_mat = new double[(_SIZE_) * (_SIZE_)];

	for (int i = 0; i < _SIZE_; ++i) {
		for (int j = 0; j < _SIZE_; ++j) {
			A_mat[(i * (_SIZE_)) + j] = ((double) ((i * (_SIZE_) + 1) + (j + 1))) / ((double) (_SIZE_));
			B_mat[(i * (_SIZE_)) + j] = ((double) ((i + 1) - (j * (_SIZE_) + 1))) / ((double) (_SIZE_));
		}
	}
}

void run_matadd(void)
{
	double a;
	double b;
	double *d_A;
	double *d_B;
	double *d_S;
	hipblasHandle_t handle;

	PERR(hipMalloc(&d_A, (_SIZE_) * (_SIZE_) * sizeof(double)));
	PERR(hipMalloc(&d_B, (_SIZE_) * (_SIZE_) * sizeof(double)));
	PERR(hipMalloc(&d_S, (_SIZE_) * (_SIZE_) * sizeof(double)));
	PSTAT(hipblasCreate(&handle));
	PSTAT(hipblasSetMatrix(_SIZE_, _SIZE_, sizeof(double), A_mat, _SIZE_, d_A, _SIZE_));
	PSTAT(hipblasSetMatrix(_SIZE_, _SIZE_, sizeof(double), B_mat, _SIZE_, d_B, _SIZE_));
	a = 1.0;
	b = 1.0;

	PSTAT(hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, _SIZE_, _SIZE_, &a, d_A, _SIZE_, &b, d_B, _SIZE_, d_S, _SIZE_));

	PSTAT(hipblasGetMatrix(_SIZE_, _SIZE_, sizeof(double), d_S, _SIZE_, S_mat, _SIZE_));
	hipblasDestroy(handle);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_S);
}

void post_matadd(void)
{
	delete[] A_mat;
	delete[] B_mat;
	delete[] S_mat;
}
