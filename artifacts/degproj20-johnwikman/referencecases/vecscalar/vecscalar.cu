#include "hip/hip_runtime.h"
/**
 * @file vecscalar.cu
 *
 * CUDA (cuBLAS) implementation of s = a*x
 */

#include <cstdlib>
#include <cmath>
#include <iostream>

#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "../benchmark_suite.hpp"

#ifndef _SIZE_
#error The _SIZE_ constant is not defined!
#endif

void pre_vecscalar(void);
void run_vecscalarcublas(void);
void run_myvecscalar(void);
void post_vecscalar(void);

#define PERR(a) p_cudaerr(#a, a)
#define PSTAT(a) p_cublasstat(#a, a)

static void p_cudaerr(const char *msg, hipError_t err)
{
    if (err != hipSuccess)
    	std::cerr << "cudaerr: " << msg << ": " << hipGetErrorString(err) << std::endl;
}

static void p_cublasstat(const char *msg, hipblasStatus_t stat)
{
#   define p_case(v) \
	       case v: \
           std::cerr << "cublasstat: " << msg << " returned " << #v << std::endl; \
           break

	switch (stat) {
	p_case(HIPBLAS_STATUS_NOT_INITIALIZED);
	p_case(HIPBLAS_STATUS_ALLOC_FAILED);
	p_case(HIPBLAS_STATUS_INVALID_VALUE);
	p_case(HIPBLAS_STATUS_ARCH_MISMATCH);
	p_case(HIPBLAS_STATUS_MAPPING_ERROR);
	p_case(HIPBLAS_STATUS_EXECUTION_FAILED);
	p_case(HIPBLAS_STATUS_INTERNAL_ERROR);
	p_case(HIPBLAS_STATUS_NOT_SUPPORTED);
	p_case(HIPBLAS_STATUS_UNKNOWN);
	default:
		break;
	}
#undef p_case
}

static double a_scalar;
static double *x_arr = NULL;
static double *s_arr = NULL;

int main(void)
{
#ifdef USE_CUSTOM
	BENCHMARK(pre_vecscalar, run_myvecscalar, post_vecscalar);
#else
	BENCHMARK(pre_vecscalar, run_vecscalarcublas, post_vecscalar);
#endif
	return 0;
}

void pre_vecscalar(void)
{
	s_arr = new double[_SIZE_];
	x_arr = new double[_SIZE_];
	for (int i = 0; i < _SIZE_; ++i) {
		x_arr[i] = ((double) (i % 101)) / (43.8);
	}
	a_scalar = 11.1;
}

void post_vecscalar(void)
{
	delete[] x_arr;
	delete[] s_arr;
}

void run_vecscalarcublas(void)
{
	double a;
	double *d_x;
	hipblasHandle_t handle;

	PERR(hipMalloc(&d_x, (_SIZE_) * sizeof(double)));
	PSTAT(hipblasCreate(&handle));
	PSTAT(hipblasSetVector(_SIZE_, sizeof(double), x_arr, 1, d_x, 1));
	a = a_scalar;

	PSTAT(hipblasDscal(handle, _SIZE_, &a, d_x, 1));

	PSTAT(hipblasGetVector(_SIZE_, sizeof(double), d_x, 1, s_arr, 1));
	hipblasDestroy(handle);
	hipFree(d_x);
}


// My own implementation
__global__ void my_vecscalar(int n, double a, double *x)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < n)
		x[i] = a * x[i];
}

void run_myvecscalar(void)
{
	double a;
	double *d_x;
	int threadsPerBlock;
	int numBlocks;

	PERR(hipDeviceGetAttribute(&threadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, 0));
	numBlocks = ((_SIZE_) + threadsPerBlock - 1) / threadsPerBlock;

	PERR(hipMalloc(&d_x, (_SIZE_) * sizeof(double)));
	PERR(hipMemcpy(d_x, x_arr, (_SIZE_) * sizeof(double), hipMemcpyHostToDevice));
	a = a_scalar;

	my_vecscalar<<<numBlocks,threadsPerBlock>>>(_SIZE_, a, d_x);
	PERR(hipDeviceSynchronize());

	PERR(hipMemcpy(s_arr, d_x, (_SIZE_) * sizeof(double), hipMemcpyDeviceToHost));
	hipFree(d_x);
}
