/**
 * @file saxpy.cu
 *
 * CUDA implementation of S = a*X + Y
 */

#include <cstdlib>
#include <cmath>
#include <iostream>

#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "../benchmark_suite.hpp"

#ifndef _SIZE_
#error The _SIZE_ constant is not defined!
#endif

void pre_saxpy(void);
void run_saxpy(void);
void post_saxpy(void);

static void p_cudaerr(const char *msg, hipError_t err)
{
    if (err != hipSuccess)
    	std::cerr << "cudaerr: " << msg << ": " << hipGetErrorString(err) << std::endl;
}

static void p_cublasstat(const char *msg, hipblasStatus_t stat)
{
#   define p_case(v) \
	       case v: \
           std::cerr << "cublasstat: " << msg << " returned " << #v << std::endl; \
           break

	switch (stat) {
	p_case(HIPBLAS_STATUS_NOT_INITIALIZED);
	p_case(HIPBLAS_STATUS_ALLOC_FAILED);
	p_case(HIPBLAS_STATUS_INVALID_VALUE);
	p_case(HIPBLAS_STATUS_ARCH_MISMATCH);
	p_case(HIPBLAS_STATUS_MAPPING_ERROR);
	p_case(HIPBLAS_STATUS_EXECUTION_FAILED);
	p_case(HIPBLAS_STATUS_INTERNAL_ERROR);
	p_case(HIPBLAS_STATUS_NOT_SUPPORTED);
	p_case(HIPBLAS_STATUS_UNKNOWN);
	default:
		break;
	}
#undef p_case
}

static double a_scalar;
static double *s_arr = NULL;
static double *x_arr = NULL;
static double *y_arr = NULL;

int main(void)
{
	BENCHMARK(pre_saxpy, run_saxpy, post_saxpy);
	return 0;
}

void pre_saxpy(void)
{
	s_arr = new double[_SIZE_];
	x_arr = new double[_SIZE_];
	y_arr = new double[_SIZE_];
	for (int i = 0; i < _SIZE_; ++i) {
		x_arr[i] = ((double) (i % 101)) / (43.8);
		y_arr[i] = ((double) (i % 103)) / (27.5);
	}
	a_scalar = 11.1;
}

void post_saxpy(void)
{
	delete[] s_arr;
	delete[] x_arr;
	delete[] y_arr;
}

void run_saxpy(void)
{
#define PERR(a) p_cudaerr(#a, a)
#define PSTAT(a) p_cublasstat(#a, a)
	double a;
	double *d_x;
	double *d_y;
	hipblasHandle_t handle;

	PERR(hipMalloc(&d_x, (_SIZE_) * sizeof(double)));
	PERR(hipMalloc(&d_y, (_SIZE_) * sizeof(double)));
	PSTAT(hipblasCreate(&handle));
	PSTAT(hipblasSetVector(_SIZE_, sizeof(double), x_arr, 1, d_x, 1));
	PSTAT(hipblasSetVector(_SIZE_, sizeof(double), y_arr, 1, d_y, 1));
	a = a_scalar;

	PSTAT(hipblasDaxpy(handle, _SIZE_, &a, d_x, 1, d_y, 1));

	PSTAT(hipblasGetVector(_SIZE_, sizeof(double), d_y, 1, s_arr, 1));
	hipblasDestroy(handle);
	hipFree(d_x);
	hipFree(d_y);
}
