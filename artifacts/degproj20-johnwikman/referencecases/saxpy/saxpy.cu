#include "hip/hip_runtime.h"
/**
 * @file saxpy.cu
 *
 * CUDA implementation of S = a*X + Y
 *
 * Note to self: For some reason the cudaMemcpys are twice as slow as when
 * running to MCore version, the kernel time is about the same. Include this in
 * the discussion and run nvprof to look at the kernel times.
 */

#include <cstdlib>
#include <cmath>
#include <iostream>

#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "../benchmark_suite.hpp"

#ifndef _SIZE_
#error The _SIZE_ constant is not defined!
#endif

void pre_saxpy(void);
void run_saxpycublas(void);
void run_saxpyexample(void);
void run_mysaxpy(void);
void post_saxpy(void);

#define PERR(a) p_cudaerr(#a, a)
#define PSTAT(a) p_cublasstat(#a, a)

static void p_cudaerr(const char *msg, hipError_t err)
{
    if (err != hipSuccess)
    	std::cerr << "cudaerr: " << msg << ": " << hipGetErrorString(err) << std::endl;
}

static void p_cublasstat(const char *msg, hipblasStatus_t stat)
{
#   define p_case(v) \
	       case v: \
           std::cerr << "cublasstat: " << msg << " returned " << #v << std::endl; \
           break

	switch (stat) {
	p_case(HIPBLAS_STATUS_NOT_INITIALIZED);
	p_case(HIPBLAS_STATUS_ALLOC_FAILED);
	p_case(HIPBLAS_STATUS_INVALID_VALUE);
	p_case(HIPBLAS_STATUS_ARCH_MISMATCH);
	p_case(HIPBLAS_STATUS_MAPPING_ERROR);
	p_case(HIPBLAS_STATUS_EXECUTION_FAILED);
	p_case(HIPBLAS_STATUS_INTERNAL_ERROR);
	p_case(HIPBLAS_STATUS_NOT_SUPPORTED);
	p_case(HIPBLAS_STATUS_UNKNOWN);
	default:
		break;
	}
#undef p_case
}

static double a_scalar;
static double *s_arr = NULL;
static double *x_arr = NULL;
static double *y_arr = NULL;

int main(void)
{
#ifdef USE_CUSTOM
	BENCHMARK(pre_saxpy, run_mysaxpy, post_saxpy);
#elif defined(USE_EXAMPLE)
	BENCHMARK(pre_saxpy, run_saxpyexample, post_saxpy);
#else
	BENCHMARK(pre_saxpy, run_saxpycublas, post_saxpy);
#endif
	return 0;
}

void pre_saxpy(void)
{
	s_arr = new double[_SIZE_];
	x_arr = new double[_SIZE_];
	y_arr = new double[_SIZE_];
	for (int i = 0; i < _SIZE_; ++i) {
		x_arr[i] = ((double) (i % 101)) / (43.8);
		y_arr[i] = ((double) (i % 103)) / (27.5);
	}
	a_scalar = 11.1;
}

void post_saxpy(void)
{
	delete[] s_arr;
	delete[] x_arr;
	delete[] y_arr;
}

void run_saxpycublas(void)
{
	double a;
	double *d_x;
	double *d_y;
	hipblasHandle_t handle;

	PERR(hipMalloc(&d_x, (_SIZE_) * sizeof(double)));
	PERR(hipMalloc(&d_y, (_SIZE_) * sizeof(double)));
	PSTAT(hipblasCreate(&handle));
	PSTAT(hipblasSetVector(_SIZE_, sizeof(double), x_arr, 1, d_x, 1));
	PSTAT(hipblasSetVector(_SIZE_, sizeof(double), y_arr, 1, d_y, 1));
	a = a_scalar;

	PSTAT(hipblasDaxpy(handle, _SIZE_, &a, d_x, 1, d_y, 1));

	PSTAT(hipblasGetVector(_SIZE_, sizeof(double), d_y, 1, s_arr, 1));
	hipblasDestroy(handle);
	hipFree(d_x);
	hipFree(d_y);
}

// Example from: https://devblogs.nvidia.com/six-ways-saxpy/
__global__
void ex_saxpy(int n, double a, double * __restrict x, double * __restrict y)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) y[i] = a*x[i] + y[i];
}

void run_saxpyexample(void)
{
	double a;
	double *d_x;
	double *d_y;
	int threadsPerBlock;
	int numBlocks;

	PERR(hipDeviceGetAttribute(&threadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, 0));
	numBlocks = ((_SIZE_) + threadsPerBlock - 1) / threadsPerBlock;

	PERR(hipMalloc(&d_x, (_SIZE_) * sizeof(double)));
	PERR(hipMalloc(&d_y, (_SIZE_) * sizeof(double)));
	PERR(hipMemcpy(d_x, x_arr, (_SIZE_) * sizeof(double), hipMemcpyHostToDevice));
	PERR(hipMemcpy(d_y, y_arr, (_SIZE_) * sizeof(double), hipMemcpyHostToDevice));
	a = a_scalar;

	ex_saxpy<<<numBlocks,threadsPerBlock>>>(_SIZE_, a, d_x, d_y);
	PERR(hipDeviceSynchronize());

	PERR(hipMemcpy(s_arr, d_y, (_SIZE_) * sizeof(double), hipMemcpyDeviceToHost));
	hipFree(d_x);
	hipFree(d_y);
}



// My own implementation
__global__
void my_saxpy(int n, double a, const double *x, const double *y, double *s)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < n)
		s[i] = (a * x[i]) + y[i];
}

void run_mysaxpy(void)
{
	double a;
	double *d_x;
	double *d_y;
	double *d_s;
	int threadsPerBlock;
	int numBlocks;

	PERR(hipDeviceGetAttribute(&threadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, 0));
	numBlocks = ((_SIZE_) + threadsPerBlock - 1) / threadsPerBlock;

	PERR(hipMalloc(&d_x, (_SIZE_) * sizeof(double)));
	PERR(hipMalloc(&d_y, (_SIZE_) * sizeof(double)));
	PERR(hipMalloc(&d_s, (_SIZE_) * sizeof(double)));
	PERR(hipMemcpy(d_x, x_arr, (_SIZE_) * sizeof(double), hipMemcpyHostToDevice));
	PERR(hipMemcpy(d_y, y_arr, (_SIZE_) * sizeof(double), hipMemcpyHostToDevice));
	a = a_scalar;

	my_saxpy<<<numBlocks,threadsPerBlock>>>(_SIZE_, a, d_x, d_y, d_s);
	PERR(hipDeviceSynchronize());

	PERR(hipMemcpy(s_arr, d_s, (_SIZE_) * sizeof(double), hipMemcpyDeviceToHost));
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_s);
}
