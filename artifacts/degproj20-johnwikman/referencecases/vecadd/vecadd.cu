#include "hip/hip_runtime.h"
/**
 * @file vecadd.cu
 *
 * CUDA (cuBLAS) implementation of s = x + y
 */

#include <cstdlib>
#include <cmath>
#include <iostream>

#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "../benchmark_suite.hpp"

#ifndef _SIZE_
#error The _SIZE_ constant is not defined!
#endif

void pre_vecadd(void);
void run_vecaddcublas(void);
void run_myvecadd(void);
void post_vecadd(void);

#define PERR(a) p_cudaerr(#a, a)
#define PSTAT(a) p_cublasstat(#a, a)

static void p_cudaerr(const char *msg, hipError_t err)
{
    if (err != hipSuccess)
    	std::cerr << "cudaerr: " << msg << ": " << hipGetErrorString(err) << std::endl;
}

static void p_cublasstat(const char *msg, hipblasStatus_t stat)
{
#   define p_case(v) \
	       case v: \
           std::cerr << "cublasstat: " << msg << " returned " << #v << std::endl; \
           break

	switch (stat) {
	p_case(HIPBLAS_STATUS_NOT_INITIALIZED);
	p_case(HIPBLAS_STATUS_ALLOC_FAILED);
	p_case(HIPBLAS_STATUS_INVALID_VALUE);
	p_case(HIPBLAS_STATUS_ARCH_MISMATCH);
	p_case(HIPBLAS_STATUS_MAPPING_ERROR);
	p_case(HIPBLAS_STATUS_EXECUTION_FAILED);
	p_case(HIPBLAS_STATUS_INTERNAL_ERROR);
	p_case(HIPBLAS_STATUS_NOT_SUPPORTED);
	p_case(HIPBLAS_STATUS_UNKNOWN);
	default:
		break;
	}
#undef p_case
}

static double *s_arr = NULL;
static double *x_arr = NULL;
static double *y_arr = NULL;

int main(void)
{
#ifdef USE_CUSTOM
	BENCHMARK(pre_vecadd, run_myvecadd, post_vecadd);
#else
	BENCHMARK(pre_vecadd, run_vecaddcublas, post_vecadd);
#endif
	return 0;
}

void pre_vecadd(void)
{
	s_arr = new double[_SIZE_];
	x_arr = new double[_SIZE_];
	y_arr = new double[_SIZE_];
	for (int i = 0; i < _SIZE_; ++i) {
		x_arr[i] = ((double) (i % 101)) / (43.8);
		y_arr[i] = ((double) (i % 103)) / (27.5);
	}
}

void post_vecadd(void)
{
	delete[] s_arr;
	delete[] x_arr;
	delete[] y_arr;
}

void run_vecaddcublas(void)
{
	double a;
	double *d_x;
	double *d_y;
	hipblasHandle_t handle;

	PERR(hipMalloc(&d_x, (_SIZE_) * sizeof(double)));
	PERR(hipMalloc(&d_y, (_SIZE_) * sizeof(double)));
	PSTAT(hipblasCreate(&handle));
	PSTAT(hipblasSetVector(_SIZE_, sizeof(double), x_arr, 1, d_x, 1));
	PSTAT(hipblasSetVector(_SIZE_, sizeof(double), y_arr, 1, d_y, 1));
	a = 1.0;

	PSTAT(hipblasDaxpy(handle, _SIZE_, &a, d_x, 1, d_y, 1));

	PSTAT(hipblasGetVector(_SIZE_, sizeof(double), d_y, 1, s_arr, 1));
	hipblasDestroy(handle);
	hipFree(d_x);
	hipFree(d_y);
}



// My own implementation
__global__
void my_vecadd(int n, const double *x, double *y)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < n)
		y[i] = x[i] + y[i];
}

void run_myvecadd(void)
{
	double *d_x;
	double *d_y;
	int threadsPerBlock;
	int numBlocks;

	PERR(hipDeviceGetAttribute(&threadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, 0));
	numBlocks = ((_SIZE_) + threadsPerBlock - 1) / threadsPerBlock;

	PERR(hipMalloc(&d_x, (_SIZE_) * sizeof(double)));
	PERR(hipMalloc(&d_y, (_SIZE_) * sizeof(double)));
	PERR(hipMemcpy(d_x, x_arr, (_SIZE_) * sizeof(double), hipMemcpyHostToDevice));
	PERR(hipMemcpy(d_y, y_arr, (_SIZE_) * sizeof(double), hipMemcpyHostToDevice));

	my_vecadd<<<numBlocks,threadsPerBlock>>>(_SIZE_, d_x, d_y);
	PERR(hipDeviceSynchronize());

	PERR(hipMemcpy(s_arr, d_y, (_SIZE_) * sizeof(double), hipMemcpyDeviceToHost));
	hipFree(d_x);
	hipFree(d_y);
}


