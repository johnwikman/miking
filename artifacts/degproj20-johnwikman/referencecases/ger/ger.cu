/**
 * @file ger.cu
 *
 * CUDA (cuBLAS) implementation of S = x*y^T + A
 */

#include <cstdlib>
#include <cmath>
#include <iostream>

#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "../benchmark_suite.hpp"

#ifndef _SIZE_
#error The _SIZE_ constant is not defined!
#endif

void pre_ger(void);
void run_ger(void);
void post_ger(void);

#define PERR(a) p_cudaerr(#a, a)
#define PSTAT(a) p_cublasstat(#a, a)

static void p_cudaerr(const char *msg, hipError_t err)
{
    if (err != hipSuccess)
    	std::cerr << "cudaerr: " << msg << ": " << hipGetErrorString(err) << std::endl;
}

static void p_cublasstat(const char *msg, hipblasStatus_t stat)
{
#   define p_case(v) \
	       case v: \
           std::cerr << "cublasstat: " << msg << " returned " << #v << std::endl; \
           break

	switch (stat) {
	p_case(HIPBLAS_STATUS_NOT_INITIALIZED);
	p_case(HIPBLAS_STATUS_ALLOC_FAILED);
	p_case(HIPBLAS_STATUS_INVALID_VALUE);
	p_case(HIPBLAS_STATUS_ARCH_MISMATCH);
	p_case(HIPBLAS_STATUS_MAPPING_ERROR);
	p_case(HIPBLAS_STATUS_EXECUTION_FAILED);
	p_case(HIPBLAS_STATUS_INTERNAL_ERROR);
	p_case(HIPBLAS_STATUS_NOT_SUPPORTED);
	p_case(HIPBLAS_STATUS_UNKNOWN);
	default:
		break;
	}
#undef p_case
}

static double *x_vec = NULL;
static double *y_vec = NULL;
static double *A_mat = NULL;
static double *S_mat = NULL;

int main(void)
{
	BENCHMARK(pre_ger, run_ger, post_ger);
	return 0;
}


void pre_ger(void)
{
	x_vec = new double[_SIZE_];
	y_vec = new double[_SIZE_];
	A_mat = new double[(_SIZE_) * (_SIZE_)];
	S_mat = new double[(_SIZE_) * (_SIZE_)];

	for (int i = 0; i < _SIZE_; ++i) {
		x_vec[i] = 1.18 - (1.0 / ((double) (i + 1)));
		y_vec[i] = 0.376 + (3.0 / ((double) (i + 1)));

		for (int j = 0; j < _SIZE_; ++j) {
			A_mat[(i * (_SIZE_)) + j] = ((double) ((i * (_SIZE_) + 1) + (j + 1))) / ((double) (_SIZE_));
		}
	}
}

void run_ger(void)
{
	double a;
	double *d_x;
	double *d_y;
	double *d_A;
	hipblasHandle_t handle;

	PERR(hipMalloc(&d_x, (_SIZE_) * sizeof(double)));
	PERR(hipMalloc(&d_y, (_SIZE_) * sizeof(double)));
	PERR(hipMalloc(&d_A, (_SIZE_) * (_SIZE_) * sizeof(double)));
	PSTAT(hipblasCreate(&handle));
	PSTAT(hipblasSetVector(_SIZE_, sizeof(double), x_vec, 1, d_x, 1));
	PSTAT(hipblasSetVector(_SIZE_, sizeof(double), y_vec, 1, d_y, 1));
	PSTAT(hipblasSetMatrix(_SIZE_, _SIZE_, sizeof(double), A_mat, _SIZE_, d_A, _SIZE_));
	a = 1.0;

	PSTAT(hipblasDger(handle, _SIZE_, _SIZE_, &a, d_x, 1, d_y, 1, d_A, _SIZE_));

	PSTAT(hipblasGetMatrix(_SIZE_, _SIZE_, sizeof(double), d_A, _SIZE_, S_mat, _SIZE_));
	hipblasDestroy(handle);
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_A);
}

void post_ger(void)
{
	delete[] x_vec;
	delete[] y_vec;
	delete[] A_mat;
	delete[] S_mat;
}
